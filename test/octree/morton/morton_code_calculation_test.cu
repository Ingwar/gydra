#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <src/octree/morton.h>

#include <test/utils/matchers/bitwise.h>
#include "matchers/morton_code.h"
#include "property/morton_code.h"
#include "utils.h"

namespace gydra {

namespace octree {

namespace morton {

namespace test {

using gydra::testing::matchers::HasNoMoreThanNSignificantBits;


class MortonCodeTest : public PropertyTestForAllPointsWithCoordinatesHavingLessThan20Bits {
};


class MortonCodeLengthTest: public PropertyTestForAllPoints {
};


PROPERTY_TEST(MortonCodeTest, compute_morton_code_should_correctly_calculate_morton_code, point) {
  const MortonCode result = compute_morton_code(point);
  ASSERT_THAT(result, IsCorrectMortonCodeForPoint(point));
}

PROPERTY_TEST(MortonCodeTest, compute_morton_code_should_return_same_result_for_points_which_have_same_first_20_bits_of_coordidates, point) {

  const unsigned int shift = 1 << 22;

  const size_t number_of_cases = 7;

  const MortonCode expected_morton_code = compute_morton_code(point);

  const uint3 shifted_point_array[number_of_cases] = {
    make_uint3(point.x + shift, point.y, point.z),
    make_uint3(point.x, point.y + shift, point.z),
    make_uint3(point.x, point.y, point.z + shift),
    make_uint3(point.x + shift, point.y + shift, point.z),
    make_uint3(point.x + shift, point.y, point.z + shift),
    make_uint3(point.x, point.y + shift, point.z + shift),
    make_uint3(point.x + shift, point.y + shift, point.z + shift)
  };

  for (size_t i = 0; i < number_of_cases; i++) {
    const uint3 shifted_point = shifted_point_array[i];
    const MortonCode morton_code_for_shifted_point = compute_morton_code(shifted_point);
    ASSERT_EQ(expected_morton_code, morton_code_for_shifted_point) << "Morton code differs for points " << point << " and " << shifted_point;
  }

}


PROPERTY_TEST(MortonCodeLengthTest, morton_code_should_be_no_longer_than_60_bits, point) {
  const MortonCode result = compute_morton_code(point);
  ASSERT_THAT(result, HasNoMoreThanNSignificantBits(60));
}

} //  namespace test

} //  namespace morton

} //  namespace octree

} //  namespace gydra
